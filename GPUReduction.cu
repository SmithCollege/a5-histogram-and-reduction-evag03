
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define SIZE 100

__global__ void Reduction(int* input, int* output, int operation) {
  // Stride is distance to the next value being
  // accumulated into the threads mapped position
  // in the partialSum[] aray
  /*
  for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
    __syncthreads();
    
    if (t % stride == 0) {
      partialSum[2*t]+= partialSum[2*t+stride]; 
    }
  } 
  */

  // Sum Operation
  if (operation == 0) {
    for (unsigned int stride = 1; stride <= SIZE; stride *= 2) {
      if (SIZE % stride == 0) {
        // output[st output[2*SIZE] = input[2*SIZE] + input[2*SIZE*stride];
      }
    // printf("Output: %d", output[stride]);
    }
  }
  printf ("Sum: %d", output[SIZE]);
}
												
int main(){
  // allocate memory
  int* input = (int*) malloc(sizeof(int) * SIZE);
  int* output = (int*) malloc(sizeof(int) * SIZE);

  // initialize inputs
  for (int i = 0; i < SIZE; i++) {
    input[i] = 1; 
  }

  //Reduction(input, output, 0);
  Reduction<<<1, SIZE>>>(input, output, 0);
  //printf("Output: %d", Reduction(input, output, 0));
  
  /*
  // check results
  for (int i = 0; i < SIZE; i++) {
    printf("%d ", output[i]);
  }
  printf("\n");
  */

  hipFree(input);
  hipFree(output);
  
  return 0;

}